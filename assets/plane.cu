#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, normal, attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(float3, n, , );
rtDeclareVariable(float3, pos, , );


RT_PROGRAM void planeIntersect(int prim) {
	float den = dot(n, ray.direction);
	if(den < 0.0f) {
		float3 hitPoint = pos-ray.origin;
		float d = dot(hitPoint, n) / den;
		//if(hitPoint >= 0) {
			if(rtPotentialIntersection(d)) {
				normal = n;
				texcoord.x = 0.0f;
				texcoord.y = 0.0f;
				texcoord.z = 0.0f;
				rtReportIntersection(0);
			}
		//}
	}
}

RT_PROGRAM void planeBounds(int, float result[6]) {
	float3 min = n+pos - 10;//- 1e-6;
	float3 max = n+pos + 10;//+ 1e-6;

	result[0] = min.x;
	result[1] = min.y;
	result[2] = min.z;
	result[3] = max.x;
	result[4] = max.y;
	result[5] = max.z;
}

