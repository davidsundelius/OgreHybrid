#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>
#include "common.h"

using namespace optix;

rtDeclareVariable(float3, normal, attribute shading_normal, ); 
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtDeclareVariable(float4, sphere, , );

RT_PROGRAM void sphereIntersect(int prim) {
	float3 center = make_float3(sphere.x, sphere.y, sphere.z);
	float radius = sphere.w;
	float3 O = ray.origin - center;
	float b = dot(O, ray.direction);
	float c = dot(O, O) - radius*radius;
	float disc = b*b - c;
	if(disc > 0.0f) {
		float sdisc = sqrtf(disc);
		float root1 = (-b - sdisc);
		bool check_second = true;
		if(rtPotentialIntersection(root1)) {
			normal = (root1*ray.direction + O) / radius;
			float3 cs = normalize(center-(root1*ray.direction + O));
			texcoord.x = 0.5-atan2(cs.z, cs.x)/(2*PI);
			texcoord.y = 0.5-2*asin(cs.y)/(2*PI);
			texcoord.z = 0.0f;
			if(rtReportIntersection(0)) {
				check_second = false;
			}
		}
		if(check_second) {
			float root2 = (-b + sdisc);
			if(rtPotentialIntersection(root2)) {
				normal = (root2*ray.direction + O) / radius;
				float3 cs = normalize(center-(root1*ray.direction + O));
				texcoord.x = 0.5-atan2(cs.z, cs.x)/(2*PI);
				texcoord.y = 0.5-2*asin(cs.y)/(2*PI);
				texcoord.z = 0.0f;
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void sphereBounds(int, float result[6]) {
	float3 cen = make_float3(sphere);
	float3 rad = make_float3(sphere.w);

	float3 min = cen - rad;
	float3 max = cen + rad;

	result[0] = min.x;
	result[1] = min.y;
	result[2] = min.z;
	result[3] = max.x;
	result[4] = max.y;
	result[5] = max.z;
}

