#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

#include "common.h"

using namespace optix;

rtDeclareVariable(float3, normal, attribute shading_normal, );
rtDeclareVariable(float3, gnormal, attribute geometric_normal, );
rtDeclareVariable(float3, texcoord, attribute texcoord, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, hit, rtIntersectionDistance, );
rtDeclareVariable(Result, res, rtPayload, );
rtDeclareVariable(ResultShadow, sRes, rtPayload, );

rtDeclareVariable(rtObject, scene, , );
rtDeclareVariable(uint, rayTypeShadow, , );
rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(float3, ambientColor, , );
rtDeclareVariable(float3, ambient, , );
rtDeclareVariable(float3, diffuse, , );
rtDeclareVariable(float3, specular, , );
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float, shininess, , );
rtDeclareVariable(float, reflexive, , );
rtDeclareVariable(float, opacity, , );
rtDeclareVariable(float, refractiveIndex, , );
rtDeclareVariable(int, isTextured, , );
rtTextureSampler<float4, 2> sampler;
rtBuffer<Light> lights;


RT_PROGRAM void closestHit() {
	float3 n = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, normal));
	float3 gn = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, gnormal));
	float3 ffn = faceforward(n, -ray.direction, gn);
	float3 hitPoint = ray.origin + hit * ray.direction;

	//Ambient
	float3 tA;
	if(isTextured) {
		tA = opacity*make_float3(tex2D(sampler, texcoord.x, texcoord.y))*ambient*0.3f;
	} else {
		tA = (1.0-reflexive)*opacity*ambient*ambientColor;
	}

	//Reflections
	float3 tRefl = make_float3(0.0f);
	if(reflexive>0.0f && res.depth<MAXDEPTH && res.importance>MINIMPORTANCE) {
		float s = fresnel_schlick(-dot(ffn, ray.direction), 5, reflexive, 0.8f);
		Result reflRes;
		reflRes.importance = res.importance*optix::luminance(make_float3(reflexive));
		reflRes.depth = res.depth+1;
		float3 rv = reflect(ray.direction, ffn);
		Ray reflRay = make_Ray(hitPoint, rv, rayTypeRadience, 0.05f, RT_DEFAULT_MAX);
		rtTrace(scene, reflRay, reflRes);
		tRefl = s * reflRes.color;
	}

	//Direct
	float3 tD = make_float3(0.0f);
	if(opacity>0.0f || res.depth>=MAXDEPTH) {
		for(int i=0; i<lights.size(); ++i) {
			Light l = lights[i];
			float3 lv = normalize(l.pos - hitPoint);
			float theta = dot(ffn, lv);
			if(theta > 0.0f) {
				//Shadows
				ResultShadow sRes;
				sRes.attenuation = 1.0f;
				float ld = length(l.pos - hitPoint);
				Ray sRay = make_Ray(hitPoint, l.pos, rayTypeShadow, 0.1f, ld);
				rtTrace(scene, sRay, sRes);
				if(sRes.attenuation > 0.0f) {
					//Diffuse
					float3 lc = l.color*sRes.attenuation;
					float3 diff;
					if(isTextured) {
						diff = make_float3(tex2D(sampler, texcoord.x, texcoord.y))*theta*lc;
					} else {
						diff = diffuse*theta*lc;
					}
					float3 spec = make_float3(0.0f);
					float3 lrv = reflect(lv, ffn);
					float phi = dot(lrv, normalize(ray.direction));
					if(phi > 0) {
						spec = specular*powf(phi,shininess)*l.color;
					}
					tD += opacity*(diff+spec)*(1.0-reflexive);
				}
			}
		}
		tD/=lights.size();
	}

	//Refractions
	float3 tRefr = make_float3(0.0f);
	if(opacity<1.0f && res.depth<MAXDEPTH && res.importance>MINIMPORTANCE) {
		float3 rv;
		if(refract(rv, ray.direction, ffn, refractiveIndex)) {
			float t = dot(ray.direction, ffn);
			if(t < 0.0f)
				t = -t;
			else
				t = dot(rv, n);
			Result refrRes;
			refrRes.importance = res.importance*optix::luminance(make_float3(1.0f-opacity));
			refrRes.depth = res.depth+1;
			Ray refrRay = make_Ray(hitPoint, rv, rayTypeRadience, 0.05f, RT_DEFAULT_MAX);
			rtTrace(scene, refrRay, refrRes);
			tRefr += (1.0f-opacity)*refrRes.color;
		}
	}
	res.color = tA + tD + tRefr + tRefl + emissive;
}

RT_PROGRAM void anyHitShadow() {
	if(opacity==1.0f) {
		sRes.attenuation=0.0f;
		rtTerminateRay();
	} else {
		float3 n = normalize(normal);
		float theta = fabs(dot(n, ray.direction));
		//Caustics (very approximated)
		float ri = clamp(refractiveIndex-1.0f, 0.0f, 1.0f);
		if(ri>0.01f) {
			float exp = powf(2.0f,lerp(0.0f,9.0f,ri));
			sRes.attenuation *= powf(theta, exp)*refractiveIndex*lerp(1.0f,5.0f,ri);
		}
		//Shadow
		sRes.attenuation *= 1.0f-fresnel_schlick(theta, 5, opacity, 1);

		float luminance = optix::luminance(make_float3(sRes.attenuation));
		if(luminance < 0.01f || luminance > 1.0f) {
			rtTerminateRay();
		} else {
			rtIgnoreIntersection();
		}
	}
}
