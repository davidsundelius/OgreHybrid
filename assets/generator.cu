#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

#include "common.h"

using namespace optix;

#define epsilon 0.001f

rtDeclareVariable(uint2 , index    , rtLaunchIndex, );

rtDeclareVariable(uint, rayTypeRadience, , );
rtDeclareVariable(float3, camPos   , , );
rtDeclareVariable(float3, u, , );
rtDeclareVariable(float3, v, , );
rtDeclareVariable(float3, w, , );

rtDeclareVariable(uint , numSamples, , );

rtDeclareVariable(rtObject, scene, , );
rtDeclareVariable(int, skipMask, , );
rtTextureSampler<float, 2> mask;
rtBuffer<float4, 2> outBuffer;


RT_PROGRAM void generate() {
	float2 sc = make_float2(index) / make_float2(outBuffer.size());
	if(tex2D(mask, sc.x, sc.y) != 0.0f || skipMask) {
		float2 d = sc * 2.0f - 1.0f;
		float3 orgin = camPos;
		float3 direction = w + d.x*u - d.y*v;
		Result res;
		float3 color = make_float3(0.0f);
		float3 tmpDir;
		for(int i=0;i<numSamples;i++) {
			if(i==0) tmpDir = direction;
			else if(i==1) tmpDir = direction+epsilon*u+epsilon*v;
			else if(i==2) tmpDir = direction-epsilon*u+epsilon*v;
			else if(i==3) tmpDir = direction-epsilon*u-epsilon*v;
			else tmpDir = direction+epsilon*u-epsilon*v;
			Ray r = make_Ray(orgin, tmpDir, rayTypeRadience, 0.005f, RT_DEFAULT_MAX);
			res.depth = 0;
			res.importance = 1.0f;
			rtTrace(scene, r, res);
			color+=res.color;
		}
		color/=numSamples;
		outBuffer[index] = make_float4(color,1.0f);
	} else {
		outBuffer[index] = make_float4(0.0f, 0.0f, 0.0f, 0.0f);
	}
}


RT_PROGRAM void exception() {
	const unsigned int code = rtGetExceptionCode();
	if( code == RT_EXCEPTION_STACK_OVERFLOW )
		outBuffer[index] = make_float4(1.0f,0.0f,1.0f,1.0f);
	else
		rtPrintExceptionDetails();
}