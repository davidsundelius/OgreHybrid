#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>
#include <Optix/optixu/optixu_aabb_namespace.h>

using namespace optix;

rtDeclareVariable(float3, normal, attribute normal, ); 
rtDeclareVariable(Ray, ray, rtCurrentRay, );

rtBuffer<float3> vertexBuffer;     
rtBuffer<float3> normalBuffer;
//rtBuffer<float2> texcoordBuffer;
rtBuffer<int3>   vIndexBuffer;
rtBuffer<int3>   nIndexBuffer;
//rtBuffer<int3>   tIndexBuffer;

//rtBuffer<uint>   materialBuffer;
rtDeclareVariable(float2, texcoord, attribute texcoord, ); 

RT_PROGRAM void meshIntersect(int prim) {
  int3 vid = vIndexBuffer[prim];
  float3 p0 = vertexBuffer[vid.x];
  float3 p1 = vertexBuffer[vid.y];
  float3 p2 = vertexBuffer[vid.z];

  // Intersect ray with triangle
  float3 n;
  float  t, beta, gamma;
  if(intersect_triangle(ray, p0, p1, p2, n, t, beta, gamma)) {
    if(rtPotentialIntersection(t)) {
      //int3 nid = nIndexBuffer[prim];
      //if (normalBuffer.size() == 0 || nid.x < 0 || nid.y < 0 || nid.z < 0) {
        normal = normalize(n);
      /*} else {
        float3 n0 = normalBuffer[nid.x];
        float3 n1 = normalBuffer[nid.y];
        float3 n2 = normalBuffer[nid.z];
        normal = normalize(n1*beta + n2*gamma + n0*(1.0f-beta-gamma));
      }*/

	  texcoord = make_float2(0.0f, 0.0f);

      /*int3 tid = tIndexBuffer[prim];
      if (texcoordBuffer.size() == 0 || tid.x < 0 || tid.y < 0 || tid.z < 0) {
        
      } else {
        float2 t0 = texcoordBuffer[tid.x];
        float2 t1 = texcoordBuffer[tid.y];
        float2 t2 = texcoordBuffer[tid.z];
        texcoord = make_float3(t1*beta + t2*gamma + t0*(1.0f-beta-gamma));
      }*/
      rtReportIntersection(0); //material_buffer[primIdx]
    }
  }
}

RT_PROGRAM void meshBounds (int prim, float result[6]) {  
  const int3 vid = vIndexBuffer[prim];
  const float3 v0   = vertexBuffer[vid.x];
  const float3 v1   = vertexBuffer[vid.y];
  const float3 v2   = vertexBuffer[vid.z];
  const float area = length(cross(v1-v0, v2-v0));

  optix::Aabb* aabb = (optix::Aabb*)result;
  
  if(area > 0.0f && !isinf(area)) {
    aabb->m_min = fminf(fminf(v0, v1), v2);
    aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
  } else {
    aabb->invalidate();
  }
}
