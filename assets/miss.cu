#include "hip/hip_runtime.h"
#include <Optix/optix.h>
#include <Optix/optixu/optixu_math_namespace.h>

#include "common.h"

using namespace optix;

rtDeclareVariable(Result, res, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(uint2 , index, rtLaunchIndex, );

rtBuffer<float4, 2> outBuffer;

rtTextureSampler<float4, 2> tex;
//rtTextureSampler<float4, hipTextureTypeCubemap> tex2; //Kepler required...
rtDeclareVariable(float3, up, , );

RT_PROGRAM void miss(void) {
	float t = max(dot(ray.direction, up), 0.0f);
	float theta = atan2f(ray.direction.x, ray.direction.z);
	float phi   = M_PIf * 0.5f - acosf(ray.direction.y);
	float u     = (theta) * (0.5f * M_1_PIf);
	float v     = 0.5f * (1.0f + sin(phi));
	//texCubemap(tex2, u, u, u);
	res.color = make_float3(tex2D(tex, -u, -v));
}
